#include "hip/hip_runtime.h"
#include <cmath>
#include "kernels.h"
#include "../main.h"

#define BLOCK_SIZE 32

__global__ void sepia(const unsigned char* data, unsigned char* out, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int rgb_offset = y * width + x;

    if (x > width && y > height && rgb_offset > (width * height)) {
        return;
    }

    int r = (data[rgb_offset * 3] * 0.393f) + (data[rgb_offset * 3 + 1] * 0.769) + (data[rgb_offset * 3 + 2] * 0.189f);
    int g = (data[rgb_offset * 3] * 0.349f) + (data[rgb_offset * 3 + 1] * 0.686) + (data[rgb_offset * 3 + 2] * 0.168f);
    int b = (data[rgb_offset * 3] * 0.272f) + (data[rgb_offset * 3 + 1] * 0.534) + (data[rgb_offset * 3 + 2] * 0.131f);

    out[rgb_offset   *   3] = r > 255 ? 255 : r;
    out[rgb_offset * 3 + 1] = g > 255 ? 255 : g;
    out[rgb_offset * 3 + 2] = b > 255 ? 255 : b;
}

__global__ void negative(unsigned char* data, unsigned char* out, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int rgb_offset = y * width + x;

    if (x > width && y > height && rgb_offset > (width * height)) {
        return;
    }

    out[rgb_offset   *   3] = 255 - data[rgb_offset   *   3];
    out[rgb_offset * 3 + 1] = 255 - data[rgb_offset * 3 + 1];
    out[rgb_offset * 3 + 2] = 255 - data[rgb_offset * 3 + 2];
}

__global__ void grayscale(unsigned char* data, unsigned char* out, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int rgb_offset = y * width + x;

    if (x > width && y > height && rgb_offset > (width * height)) {
        return;
    }

    int gray = (data[rgb_offset * 3] * 0.2126f) + (data[rgb_offset * 3 + 1] * 0.7152f) + (data[rgb_offset * 3 + 2] * 0.0722f);
    out[rgb_offset   *   3] = gray;
    out[rgb_offset * 3 + 1] = gray;
    out[rgb_offset * 3 + 2] = gray;
}

__global__ void nashville(unsigned char* data, unsigned char* out, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int rgb_offset = y * width + x;

    if (x > width && y > height && rgb_offset > (width * height)) {
        return;
    }

    // Funciona como o efeito Levels (em um Photoshop/GIMP)
    // Limita o valor do output das cores
    out[rgb_offset   *   3] = data[rgb_offset   *   3] < 19  ? 19  : data[rgb_offset   *   3];
    out[rgb_offset * 3 + 1] = data[rgb_offset * 3 + 1] < 39  ? 39  : data[rgb_offset * 3 + 1];
    out[rgb_offset * 3 + 2] = data[rgb_offset * 3 + 2] < 137 ? 137 : data[rgb_offset * 3 + 2];
}

__device__ inline unsigned char get_intensity(const unsigned char *image, int row, int col,
                                             int channel, int height, int width, int cpp)
{
    if (col < 0 || col >= width)
        return 0;
    if (row < 0 || row >= height)
        return 0;
    
    /* Retorna o valor do pixel */
    return image[(row * width + col) * cpp + channel];
}

__global__ void sharpen(const unsigned char *data, unsigned char *out, const int width, const int height, const int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        for (int c = 0; c < channels; c++)
        {
            /*              x   y   z
             * Filtro = a | 0  -1   0 |
             *          b |-1   5  -1 |
             *          c | 0  -1   0 |
		     *
             * Fonte: https://setosa.io/ev/image-kernels/
             */

            unsigned char ay = get_intensity(data, y - 1, x, c, height, width, channels);
            unsigned char bx = get_intensity(data, y, x - 1, c, height, width, channels);
            unsigned char by = get_intensity(data, y, x, c, height, width, channels);
            unsigned char bz = get_intensity(data, y, x + 1, c, height, width, channels);
            unsigned char cy = get_intensity(data, y + 1, x, c, height, width, channels);

            short res = (5 * by - ay - bx - bz - cy);
            res = min(res, 255);
            res = max(res, 0);
            out[(y * width + x) * channels + c] = (unsigned char) res;
        }
    }
}

void run_kernels(const unsigned char* input_image, unsigned char* output_image, unsigned width, unsigned height)
{
	/* Cria as variáveis, para serem usadas na placa de video */
    unsigned char* dev_input;
    unsigned char* dev_output;

	/* Aloca as variáveis criadas anteriormente na GPU */
    hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char));
    hipMalloc( (void**) &dev_output, width*height*3*sizeof(unsigned char));

	/* Copia os dados da imagem original para a GPU */
    hipMemcpy( dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice );

	/* Calcula os blocos e as threads a serem usadas */
    const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 gridSize((width + blockSize.x - 1)/blockSize.x, (height + blockSize.y - 1)/blockSize.y);

	// Filtros:

    // // Grayscale
    grayscale<<<gridSize, blockSize>>>(dev_input, dev_output, width, height);
    hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);
    write_file("out/grayscale.png", output_image, width, height);

    // // Negative
    negative<<<gridSize, blockSize>>>(dev_input, dev_output, width, height);
    hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);
    write_file("out/negative.png", output_image, width, height);

    // // Sepia
    sepia<<<gridSize, blockSize>>>(dev_input, dev_output, width, height);
    hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);
    write_file("out/sepia.png", output_image, width, height);

    // // Nashville
    nashville<<<gridSize, blockSize>>>(dev_input, dev_output, width, height);
    hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);
    write_file("out/nashville.png", output_image, width, height);

    // Sharpen
    sharpen<<<gridSize, blockSize>>>(dev_input, dev_output, width, height, 3);
    hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);
    write_file("out/sharpen.png", output_image, width, height);

    hipFree(dev_input);
    hipFree(dev_output);
}

